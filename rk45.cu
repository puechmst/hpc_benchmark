#include <hip/hip_runtime.h>

#include <math.h>

#include <iostream>
#include <random>

const float A11 = 1.0 / 4.0;
const float A21 = 3.0 / 32.0;
const float A22 = 9.0 / 32.0;
const float A31 = 1932.0 / 2197.0;
const float A32 = -7200.0 / 2197.0;
const float A33 = 7296.0 / 2197.0;
const float A41 = 439.0 / 216.0;
const float A42 = -8.0;
const float A43 = 3680.0 / 513.0;
const float A44 = -845.0 / 4104.0;
const float A51 = -8.0 / 27.0;
const float A52 = 2.0;
const float A53 = -3544.0 / 2565.0;
const float A54 = 1859.0 / 4104.0;
const float A55 = -11.0 / 40.0;

const float B11 = 25.0 / 216.0;
const float B12 = 0.0;
const float B13 = 1408.0 / 2565.0;
const float B14 = 2197.0 / 4101.0;
const float B15 = -1.0 / 5.0;

const float B21 = 16.0 / 135.0;
const float B22 = 0.0;
const float B23 = 6656.0 / 12825.0;
const float B24 = 28561.0 / 56430.0;
const float B25 = -9.0 / 50.0;
const float B26 = 2.0 / 55.0;

const float C2 = 1.0 / 4.0;
const float C3 = 3.0 / 8.0;
const float C4 = 12.0 / 13.0;
const float C5 = 1.0;
const float C6 = 1.0 / 2.0;

// the dimension of the state space must be small enough to fit into local registers (255).
// static definition allows the compiler to unroll loops

#define STATE_DIM (10)

#define BSIZE (100)
#define NEQ (100 * BSIZE)

__device__ void sysdyn(float t, float *y, float *yp)
{
    // solution: y = tan(t)
    for (int i = 0; i < STATE_DIM; i++)
        yp[i] = 1.0f + y[i] * y[i];
}

__global__ void rk45(float t, float *y, float *err, float step)
{
    // arrays are normally stored in registers unless STATE_DIM is too large
    // the -Xptvas -v option in CmakeLists.txt dumps true usage.
    float yy[STATE_DIM], cur[STATE_DIM], k1[STATE_DIM], k2[STATE_DIM], k3[STATE_DIM], k4[STATE_DIM], k5[STATE_DIM], k6[STATE_DIM];
    float e;
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * STATE_DIM;
    // load local data
    for (int i = 0; i < STATE_DIM; i++)
        yy[i] = y[idx + i];
    sysdyn(t, yy, k1);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + step * A11 * k1[i];
    sysdyn(t + step * C2, cur, k2);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + step * (A21 * k1[i] + A22 * k2[i]);
    sysdyn(t + step * C3, cur, k3);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + step * (A31 * k1[i] + A32 * k2[i] + A33 * k3[i]);
    sysdyn(t + step * C4, cur, k4);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + step * (A41 * k1[i] + A42 * k2[i] + A43 * k3[i] + A44 * k4[i]);
    sysdyn(t + step * C5, cur, k5);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + step * (A51 * k1[i] + A52 * k2[i] + A53 * k3[i] + A54 * k4[i] + A55 * k5[i]);
    sysdyn(t + step * C6, cur, k6);
    // get new state and estimate error
    e = 0.0;
    for (int i = 0; i < STATE_DIM; i++)
    {
        // It is tempting to use the higher order approximation, but the predicted error is computed for the lower one,
        // and so is the optimal step.
        y[i + idx] = yy[i] + step * (B11 * k1[i] + B12 * k2[i] + B13 * k3[i] + B14 * k4[i] + B15 * k5[i]);
        e += step * fabs((B11 - B21) * k1[i] + (B12 - B22) * k2[i] + (B13 - B23) * k3[i] + (B14 - B24) * k4[i] + (B15 - B25) * k5[i] - B26 * k6[i]);
    }
    err[blockIdx.x * blockDim.x + threadIdx.x] = e;
}

int main(int argc, char *argv[])
{
    float *y, *err, *ys;
    float *dy, *derr;
    float step = 0.1;
    std::mt19937 gen;
    std::uniform_real_distribution<float> dis(0, 0.1);
    int nb = (NEQ + BSIZE - 1) / BSIZE;
    y = new float[NEQ * STATE_DIM];
    ys = new float[NEQ * STATE_DIM];
    err = new float[NEQ];
    // populate state randomly and init error
    for (int i = 0; i < NEQ; i++)
    {
        for (int j = 0; j < STATE_DIM; j++)
            y[i * STATE_DIM + j] = dis(gen);
    }
    hipMalloc(&dy, NEQ * STATE_DIM * sizeof(float));
    hipMalloc(&derr, NEQ * sizeof(float));
    // copy to device
    hipMemcpy(dy, y, NEQ * STATE_DIM * sizeof(float), hipMemcpyHostToDevice);
    // linear grid
    rk45<<<nb, BSIZE>>>(0.0, dy, derr, step);
    hipDeviceSynchronize();
    hipMemcpy(ys, dy, sizeof(float) * NEQ * STATE_DIM, hipMemcpyDeviceToHost);
    hipMemcpy(err, derr, sizeof(float) * NEQ, hipMemcpyDeviceToHost);
    hipFree(derr);
    hipFree(dy);
    float yt;
    float te;
    float max_err = 0.0;
    float max_pred_err = 0.0;
    for (int i = 0; i < NEQ; i++)
    {
        te = 0.0;
        for (int j = 0; j < STATE_DIM; j++)
        {
            yt = tan(step + atan(y[i * STATE_DIM + j]));
            te += abs(yt - ys[i * STATE_DIM + j]);
           
        }
        if (te > max_err)
                max_err = te;
        if (abs(err[i]) > max_pred_err)
            max_pred_err = abs(err[i]);
    }
    std::cout << "max error: " << max_err << " max predicted error: " << max_pred_err << std::endl;
    delete[] y;
    delete[] err;
}