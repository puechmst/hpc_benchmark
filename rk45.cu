#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <random>
#include <type_traits>

__device__ const float A11 = 1.0f / 4.0f;
__device__ const float A21 = 3.0f / 32.0f;
__device__ const float A22 = 9.0f / 32.0f;
__device__ const float A31 = 1932.0f / 2197.0f;
__device__ const float A32 = -7200.0f / 2197.0f;
__device__ const float A33 = 7296.0f / 2197.0f;
__device__ const float A41 = 439.0f / 216.0f;
__device__ const float A42 = -8.0f;
__device__ const float A43 = 3680.0f / 513.0f;
__device__ const float A44 = -845.0f / 4104.0f;
__device__ const float A51 = -8.0f / 27.0f;
__device__ const float A52 = 2.0f;
__device__ const float A53 = -3544.0f / 2565.0f;
__device__ const float A54 = 1859.0f / 4104.0f;
__device__ const float A55 = -11.0f / 40.0f;

__device__ const float B11 = 25.0f / 216.0f;
__device__ const float B12 = 0.0f;
__device__ const float B13 = 1408.0f / 2565.0f;
__device__ const float B14 = 2197.0f / 4101.0f;
__device__ const float B15 = -1.0f / 5.0f;

__device__ const float B21 = 16.0f / 135.0f;
__device__ const float B22 = 0.0f;
__device__ const float B23 = 6656.0f / 12825.0f;
__device__ const float B24 = 28561.0f / 56430.0f;
__device__ const float B25 = -9.0f / 50.0f;
__device__ const float B26 = 2.0f / 55.0f;

__device__ const float C2 = 1.0f / 4.0f;
__device__ const float C3 = 3.0f / 8.0f;
__device__ const float C4 = 12.0f / 13.0f;
__device__ const float C5 = 1.0f;
__device__ const float C6 = 1.0f / 2.0f;

// the dimension of the state space must be small enough to fit into local registers (255).
// static definition allows the compiler to unroll loops
// test

#define STATE_DIM (10)

#define BSIZE (200)
#define NEQ (1000 * BSIZE)


// __device__ void sysdyn(float t, float *y, float *yp)
// {
//     // solution: y = tan(t)
//     for (int i = 0; i < STATE_DIM; i++)
//         yp[i] = 1.0f + y[i] * y[i];
// }

struct ode_def {
     __device__ virtual void operator()(float t, float *y, float *yp) = 0;
     __device__ virtual float getTol() = 0;
};

struct my_test: public ode_def {
    const float tol = 1e-5;
    __device__ void operator()(float t, float *y, float *yp) {
        for (int i = 0; i < STATE_DIM; i++)
            yp[i] = 1.0f + y[i] * y[i];
    }

    __device__ float getTol() { return tol; }
};

template<class T>
concept OdeObject = std::is_base_of<ode_def, T>::value;

template<OdeObject T>
__global__ 
void rk45(T ode, float t, float *y, float *err, float *step)
{
    // arrays are normally stored in registers unless STATE_DIM is too large
    // the -Xptvas -v option in CmakeLists.txt dumps true usage.
    float yy[STATE_DIM], cur[STATE_DIM], k1[STATE_DIM], k2[STATE_DIM], k3[STATE_DIM], k4[STATE_DIM], k5[STATE_DIM], k6[STATE_DIM];
    float e;
    float h;
    int ide = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = ide * STATE_DIM;
    // load local data
    h = step[ide];
    for (int i = 0; i < STATE_DIM; i++)
        yy[i] = y[idx + i];
    ode(t, yy, k1);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * A11 * k1[i];
    ode(t + h * C2, cur, k2);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * (A21 * k1[i] + A22 * k2[i]);
    ode(t + h * C3, cur, k3);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * (A31 * k1[i] + A32 * k2[i] + A33 * k3[i]);
    ode(t + h * C4, cur, k4);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * (A41 * k1[i] + A42 * k2[i] + A43 * k3[i] + A44 * k4[i]);
    ode(t + h * C5, cur, k5);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * (A51 * k1[i] + A52 * k2[i] + A53 * k3[i] + A54 * k4[i] + A55 * k5[i]);
    ode(t + h * C6, cur, k6);
    // get new state and estimate error
    e = 0.0;
    for (int i = 0; i < STATE_DIM; i++)
    {
        // It is tempting to use the higher order approximation, but the predicted error is computed for the lower one,
        // and so is the optimal h.
        y[i + idx] = yy[i] + h * (B11 * k1[i] + B12 * k2[i] + B13 * k3[i] + B14 * k4[i] + B15 * k5[i]);
        e += h * fabs((B11 - B21) * k1[i] + (B12 - B22) * k2[i] + (B13 - B23) * k3[i] + (B14 - B24) * k4[i] + (B15 - B25) * k5[i] - B26 * k6[i]);
    }
    // save error
    err[ide] = e;
    // save optimal step for tolerance
    step[ide] =  h * 0.84 * pow( (float)STATE_DIM * ode.getTol() / e , 0.25f);
}

void dump_properties(std::ofstream &of) {
    // enumerare devices
    int ndevices;
    hipDeviceProp_t prop;
    hipGetDeviceCount(&ndevices);
    for(int i = 0 ; i < ndevices ; i++) {
        of << "Device " << i <<  ":" << std::endl;
        hipGetDeviceProperties(&prop, i);
        of << "name : " << prop.name << std::endl;
        of << "arch : " << prop.major << "." << prop.minor << std::endl;
        of << "global memory : " << prop.totalGlobalMem << std::endl;
        of << "shared memory (per block) : " << prop.sharedMemPerBlock << std::endl;
        of << "registers (per block) : " << prop.regsPerBlock << std::endl;
        of << "registers (per mp) : " << prop.regsPerMultiprocessor << std::endl;
    }
}

int main(int argc, char *argv[])
{
    float *y, *err, *ys, *istep, *step;
    float *dy, *derr, *dstep;
    std::mt19937 gen;
    std::uniform_real_distribution<float> dis(0, 0.1);
    int nb = (NEQ + BSIZE - 1) / BSIZE;
    y = new float[NEQ * STATE_DIM];
    ys = new float[NEQ * STATE_DIM];
    err = new float[NEQ];
    step = new float[NEQ];
    istep = new float[NEQ];
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // file for saving results
    std::ofstream res_file("res.txt");
    // dump capabilities
    dump_properties(res_file);
    res_file << "neq : " << NEQ << std::endl;
    res_file << "dim : " << STATE_DIM << std::endl;
    res_file << "nb : " << nb << std::endl;
    // populate state randomly and init error
    for (int i = 0; i < NEQ; i++)
    {
        for (int j = 0; j < STATE_DIM; j++)
            y[i * STATE_DIM + j] = dis(gen);
        step[i] = 0.003;
        istep[i] = step[i];
    }
    hipMalloc(&dy, NEQ * STATE_DIM * sizeof(float));
    hipMalloc(&derr, NEQ * sizeof(float));
    hipMalloc(&dstep, NEQ * sizeof(float));
    // copy to device
    hipMemcpy(dy, y, NEQ * STATE_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dstep, step, NEQ * sizeof(float), hipMemcpyHostToDevice);
    // linear grid
    hipEventRecord(start);
    rk45<<<nb, BSIZE>>>(my_test(), 0.0, dy, derr, dstep);
    hipEventRecord(stop);
    //hipDeviceSynchronize();
    hipMemcpy(ys, dy, sizeof(float) * NEQ * STATE_DIM, hipMemcpyDeviceToHost);
    hipMemcpy(err, derr, sizeof(float) * NEQ, hipMemcpyDeviceToHost);
    hipMemcpy(step, dstep, sizeof(float) * NEQ, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipFree(dstep);
    hipFree(derr);
    hipFree(dy);
    float millis=0;
    hipEventElapsedTime(&millis, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    res_file << "Elapsed : " << millis << std::endl;
    float yt;
    float te;
    float max_err = 0.0;
    float max_pred_err = 0.0;
    for (int i = 0; i < NEQ; i++)
    {
        std::cout << step[i] << std::endl;
        te = 0.0;
        for (int j = 0; j < STATE_DIM; j++)
        {
            yt = tan(istep[i] + atan(y[i * STATE_DIM + j]));
            te += abs(yt - ys[i * STATE_DIM + j]);
           
        }
        if (te > max_err)
                max_err = te;
        if (abs(err[i]) > max_pred_err)
            max_pred_err = abs(err[i]);
    }
    std::cout << "max error: " << max_err << " max predicted error: " << max_pred_err << std::endl;
    delete[] y;
    delete[] err;
    delete[] step;
    delete[] istep;
    res_file.close();
    return 0;
}