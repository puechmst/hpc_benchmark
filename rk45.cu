#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <iostream>
#include <fstream>
#include <type_traits>

__device__ const float A11 = 1.0f / 4.0f;
__device__ const float A21 = 3.0f / 32.0f;
__device__ const float A22 = 9.0f / 32.0f;
__device__ const float A31 = 1932.0f / 2197.0f;
__device__ const float A32 = -7200.0f / 2197.0f;
__device__ const float A33 = 7296.0f / 2197.0f;
__device__ const float A41 = 439.0f / 216.0f;
__device__ const float A42 = -8.0f;
__device__ const float A43 = 3680.0f / 513.0f;
__device__ const float A44 = -845.0f / 4104.0f;
__device__ const float A51 = -8.0f / 27.0f;
__device__ const float A52 = 2.0f;
__device__ const float A53 = -3544.0f / 2565.0f;
__device__ const float A54 = 1859.0f / 4104.0f;
__device__ const float A55 = -11.0f / 40.0f;

__device__ const float B11 = 25.0f / 216.0f;
__device__ const float B12 = 0.0f;
__device__ const float B13 = 1408.0f / 2565.0f;
__device__ const float B14 = 2197.0f / 4101.0f;
__device__ const float B15 = -1.0f / 5.0f;

__device__ const float B21 = 16.0f / 135.0f;
__device__ const float B22 = 0.0f;
__device__ const float B23 = 6656.0f / 12825.0f;
__device__ const float B24 = 28561.0f / 56430.0f;
__device__ const float B25 = -9.0f / 50.0f;
__device__ const float B26 = 2.0f / 55.0f;

__device__ const float C2 = 1.0f / 4.0f;
__device__ const float C3 = 3.0f / 8.0f;
__device__ const float C4 = 12.0f / 13.0f;
__device__ const float C5 = 1.0f;
__device__ const float C6 = 1.0f / 2.0f;

// the dimension of the state space must be small enough to fit into local registers (255).
// static definition allows the compiler to unroll loops
// test

#define STATE_DIM (10)

#define BSIZE (200)
#define NEQ (1000 * BSIZE)


// __device__ void sysdyn(float t, float *y, float *yp)
// {
//     // solution: y = tan(t)
//     for (int i = 0; i < STATE_DIM; i++)
//         yp[i] = 1.0f + y[i] * y[i];
// }

struct ode_def {
     __device__ virtual void operator()(float t, float *y, float *yp) = 0;
     __device__ virtual float getTol() = 0;
};

struct my_test: public ode_def {
    const float tol = 1e-5;
    __device__ void operator()(float t, float *y, float *yp) {
        for (int i = 0; i < STATE_DIM; i++)
            yp[i] = 1.0f + y[i] * y[i];
    }

    __device__ float getTol() { return tol; }
};

template<class T>
concept OdeObject = std::is_base_of<ode_def, T>::value;

template<OdeObject T>
__global__ 
void rk45(T ode, float t, float *y, float *err, float *step)
{
    // arrays are normally stored in registers unless STATE_DIM is too large
    // the -Xptvas -v option in CmakeLists.txt dumps true usage.
    float yy[STATE_DIM], cur[STATE_DIM], k1[STATE_DIM], k2[STATE_DIM], k3[STATE_DIM], k4[STATE_DIM], k5[STATE_DIM], k6[STATE_DIM];
    float e;
    float h;
    int ide = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = ide * STATE_DIM;
    // load local data
    h = step[ide];
    for (int i = 0; i < STATE_DIM; i++)
        yy[i] = y[idx + i];
    ode(t, yy, k1);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * A11 * k1[i];
    ode(t + h * C2, cur, k2);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * (A21 * k1[i] + A22 * k2[i]);
    ode(t + h * C3, cur, k3);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * (A31 * k1[i] + A32 * k2[i] + A33 * k3[i]);
    ode(t + h * C4, cur, k4);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * (A41 * k1[i] + A42 * k2[i] + A43 * k3[i] + A44 * k4[i]);
    ode(t + h * C5, cur, k5);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * (A51 * k1[i] + A52 * k2[i] + A53 * k3[i] + A54 * k4[i] + A55 * k5[i]);
    ode(t + h * C6, cur, k6);
    // get new state and estimate error
    e = 0.0;
    for (int i = 0; i < STATE_DIM; i++)
    {
        // It is tempting to use the higher order approximation, but the predicted error is computed for the lower one,
        // and so is the optimal h.
        y[i + idx] = yy[i] + h * (B11 * k1[i] + B12 * k2[i] + B13 * k3[i] + B14 * k4[i] + B15 * k5[i]);
        e += h * fabs((B11 - B21) * k1[i] + (B12 - B22) * k2[i] + (B13 - B23) * k3[i] + (B14 - B24) * k4[i] + (B15 - B25) * k5[i] - B26 * k6[i]);
    }
    // save error
    err[ide] = e;
    // save optimal step for tolerance
    step[ide] =  h * 0.84 * pow( (float)STATE_DIM * ode.getTol() / e , 0.25f);
}

void dump_properties(std::ofstream &of) {
    // enumerare devices
    int ndevices;
    hipDeviceProp_t prop;
    hipGetDeviceCount(&ndevices);
    for(int i = 0 ; i < ndevices ; i++) {
        of << "Device " << i <<  ":" << std::endl;
        hipGetDeviceProperties(&prop, i);
        of << "name : " << prop.name << std::endl;
        of << "arch : " << prop.major << "." << prop.minor << std::endl;
        of << "global memory : " << prop.totalGlobalMem << std::endl;
        of << "shared memory (per block) : " << prop.sharedMemPerBlock << std::endl;
        of << "registers (per block) : " << prop.regsPerBlock << std::endl;
        of << "registers (per mp) : " << prop.regsPerMultiprocessor << std::endl;
    }
}

int main(int argc, char *argv[])
{
    float *dy, *derr, *dstep;
    thrust::host_vector<float> y(NEQ * STATE_DIM);
    thrust::host_vector<float> ys(NEQ * STATE_DIM);
    thrust::host_vector<float> err(NEQ);
    thrust::host_vector<float> step(NEQ);
    thrust::host_vector<float> istep(NEQ);
    thrust::device_vector<float> dvy(NEQ * STATE_DIM);
    thrust::device_vector<float> dverr(NEQ);
    thrust::device_vector<float> dvstep(NEQ);
    thrust::uniform_real_distribution<float> dist(0,0.1);
    thrust::default_random_engine rng(1234);

    int nb = (NEQ + BSIZE - 1) / BSIZE;
 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // file for saving results
    std::ofstream res_file("res.txt");
    // dump capabilities
    dump_properties(res_file);
    res_file << "neq : " << NEQ << std::endl;
    res_file << "dim : " << STATE_DIM << std::endl;
    res_file << "nb : " << nb << std::endl;
    // populate state randomly
    thrust::generate(y.begin(),y.end(),[&] { return dist(rng);});
    // save state
    ys = y;
    // set step
    thrust::fill(step.begin(), step.end(), 1e-2);
    dvstep = step;
    istep = step;
    // copy state to device
    dvy = y;
    dy = thrust::raw_pointer_cast(&dvy[0]);
    derr = thrust::raw_pointer_cast(&dverr[0]);
    dstep = thrust::raw_pointer_cast(&dvstep[0]);
    hipEventRecord(start);
    rk45<<<nb, BSIZE>>>(my_test(), 0.0, dy, derr, dstep);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float millis=0;
    hipEventElapsedTime(&millis, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    res_file << "Elapsed : " << millis << std::endl;
    float yt;
    float te;
    float max_err = 0.0;
    float max_pred_err = 0.0;
    // copy back from device
    step = dvstep;
    y = dvy;
    err = dverr;
    for (int i = 0; i < NEQ; i++)
    {
        std::cout << step[i] << std::endl;
        te = 0.0;
        for (int j = 0; j < STATE_DIM; j++)
        {
            yt = tan(istep[i] + atan(ys[i * STATE_DIM + j]));
            te += abs(yt - y[i * STATE_DIM + j]);
           
        }
        if (te > max_err)
                max_err = te;
        if (abs(err[i]) > max_pred_err)
            max_pred_err = abs(err[i]);
    }
    std::cout << "max error: " << max_err << " max predicted error: " << max_pred_err << std::endl;
    res_file.close();
    return 0;
}