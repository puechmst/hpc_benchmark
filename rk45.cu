#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>
#include <iostream>
#include <fstream>
#include <type_traits>

__device__ const float A11 = 1.0f / 4.0f;
__device__ const float A21 = 3.0f / 32.0f;
__device__ const float A22 = 9.0f / 32.0f;
__device__ const float A31 = 1932.0f / 2197.0f;
__device__ const float A32 = -7200.0f / 2197.0f;
__device__ const float A33 = 7296.0f / 2197.0f;
__device__ const float A41 = 439.0f / 216.0f;
__device__ const float A42 = -8.0f;
__device__ const float A43 = 3680.0f / 513.0f;
__device__ const float A44 = -845.0f / 4104.0f;
__device__ const float A51 = -8.0f / 27.0f;
__device__ const float A52 = 2.0f;
__device__ const float A53 = -3544.0f / 2565.0f;
__device__ const float A54 = 1859.0f / 4104.0f;
__device__ const float A55 = -11.0f / 40.0f;

__device__ const float B11 = 25.0f / 216.0f;
__device__ const float B12 = 0.0f;
__device__ const float B13 = 1408.0f / 2565.0f;
__device__ const float B14 = 2197.0f / 4104.0f;
__device__ const float B15 = -1.0f / 5.0f;

__device__ const float B21 = 16.0f / 135.0f;
__device__ const float B22 = 0.0f;
__device__ const float B23 = 6656.0f / 12825.0f;
__device__ const float B24 = 28561.0f / 56430.0f;
__device__ const float B25 = -9.0f / 50.0f;
__device__ const float B26 = 2.0f / 55.0f;

__device__ const float C2 = 1.0f / 4.0f;
__device__ const float C3 = 3.0f / 8.0f;
__device__ const float C4 = 12.0f / 13.0f;
__device__ const float C5 = 1.0f;
__device__ const float C6 = 1.0f / 2.0f;

// the dimension of the state space must be small enough to fit into local registers (255).
// static definition allows the compiler to unroll loops
// test

#define STATE_DIM (10)

#define BSIZE (100)
#define NEQ (1000 * BSIZE)

struct ode_def
{
    __device__ virtual void operator()(float t, float *y, float *yp) = 0;
    __device__ __host__ virtual float getATol() = 0;
    __device__ __host__ virtual float getRTol() = 0;
};

struct my_test : public ode_def
{
    const float atol = 1e-5;
    const float rtol = 1e-2;
    __device__ void operator()(float t, float *y, float *yp)
    {
        for (int i = 0; i < STATE_DIM; i++)
            yp[i] = 1.0f + y[i] * y[i];
    }

    __device__ __host__ float getATol() { return atol; }
    __device__ __host__ float getRTol() { return rtol; }
};

template <class T>
concept OdeObject = std::is_base_of<ode_def, T>::value;

template <OdeObject T>
__global__ void rk45(T ode, float *time, float *y4, float *y5, float *step)
{
    // arrays are normally stored in registers unless STATE_DIM is too large
    // the -Xptvas -v option in CmakeLists.txt dumps true usage.
    // please check that no spill memory is used.
    float yy[STATE_DIM], cur[STATE_DIM], k1[STATE_DIM], k2[STATE_DIM], k3[STATE_DIM], k4[STATE_DIM], k5[STATE_DIM], k6[STATE_DIM];
    float h, t;
    int ide = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = ide * STATE_DIM;
    // load local data
    h = step[ide];
    t = time[ide];
    for (int i = 0; i < STATE_DIM; i++)
        yy[i] = y4[idx + i];
    ode(t, yy, k1);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * A11 * k1[i];
    ode(t + h * C2, cur, k2);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * (A21 * k1[i] + A22 * k2[i]);
    ode(t + h * C3, cur, k3);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * (A31 * k1[i] + A32 * k2[i] + A33 * k3[i]);
    ode(t + h * C4, cur, k4);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * (A41 * k1[i] + A42 * k2[i] + A43 * k3[i] + A44 * k4[i]);
    ode(t + h * C5, cur, k5);
    for (int i = 0; i < STATE_DIM; i++)
        cur[i] = yy[i] + h * (A51 * k1[i] + A52 * k2[i] + A53 * k3[i] + A54 * k4[i] + A55 * k5[i]);
    ode(t + h * C6, cur, k6);
    // get new states at order 4 and 5
    for (int i = 0; i < STATE_DIM; i++)
    {
        y4[i + idx] = yy[i] + h * (B11 * k1[i] + B12 * k2[i] + B13 * k3[i] + B14 * k4[i] + B15 * k5[i]);
        y5[i + idx] = yy[i] + h * (B21 * k1[i] + B22 * k2[i] + B23 * k3[i] + B24 * k4[i] + B25 * k5[i] + B26 * k6[i]);
    }
}

void dump_properties(std::ofstream &of)
{
    // enumerare devices
    int ndevices;
    hipDeviceProp_t prop;
    hipGetDeviceCount(&ndevices);
    for (int i = 0; i < ndevices; i++)
    {
        of << "Device " << i << ":" << std::endl;
        hipGetDeviceProperties(&prop, i);
        of << "name : " << prop.name << std::endl;
        of << "arch : " << prop.major << "." << prop.minor << std::endl;
        of << "global memory : " << prop.totalGlobalMem << std::endl;
        of << "shared memory (per block) : " << prop.sharedMemPerBlock << std::endl;
        of << "registers (per block) : " << prop.regsPerBlock << std::endl;
        of << "registers (per mp) : " << prop.regsPerMultiprocessor << std::endl;
    }
}

int main(int argc, char *argv[])
{
    float *dy4, *dy5, *dtime, *dstep;
    my_test ode;
    thrust::host_vector<float> t(NEQ);
    thrust::host_vector<float> tf(NEQ);
    thrust::host_vector<float> y(NEQ * STATE_DIM);
    thrust::host_vector<float> y5(NEQ * STATE_DIM);
    thrust::host_vector<float> ys(NEQ * STATE_DIM);
    thrust::host_vector<float> step(NEQ);
    thrust::device_vector<float> dvt(NEQ);
    thrust::device_vector<float> dvy4(NEQ * STATE_DIM);
    thrust::device_vector<float> dvy5(NEQ * STATE_DIM);
    thrust::device_vector<float> dvstep(NEQ);
    thrust::uniform_real_distribution<float> dist(0, 0.1);
    thrust::default_random_engine rng(1234);

    int nb = (NEQ + BSIZE - 1) / BSIZE;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // file for saving results
    std::ofstream res_file("res.txt");
    // dump capabilities
    dump_properties(res_file);
    res_file << "neq : " << NEQ << std::endl;
    res_file << "dim : " << STATE_DIM << std::endl;
    res_file << "nb : " << nb << std::endl;
    // populate state randomly
    thrust::generate(y.begin(), y.end(), [&]
                     { return dist(rng); });
    // save state
    ys = y;
    // set time, final time and step
    thrust::fill(t.begin(), t.end(), 0.0);
    thrust::fill(tf.begin(), tf.end(), 10.0);
    thrust::fill(step.begin(), step.end(), 1e-2);

    // iterate until final time is reached
    bool is_finished = false;
    float tpe;
    float err_level, err_estimate;
    float s;
    int nstates = 0;
    while (nstates < 100 * NEQ)
    {
        // copy state to device
        dvstep = step;
        dvy4 = ys;
        dvt = t;
        dy4 = thrust::raw_pointer_cast(&dvy4[0]);
        dy5 = thrust::raw_pointer_cast(&dvy5[0]);
        dstep = thrust::raw_pointer_cast(&dvstep[0]);
        dtime = thrust::raw_pointer_cast(&dvt[0]);
        rk45<<<nb, BSIZE>>>(ode, dtime, dy4, dy5, dstep);
        hipDeviceSynchronize();
        // copy back from device
        y = dvy4;
        y5 = dvy5;
        for (int i = 0; i < NEQ; i++)
        {
            // check for termination
            if (t[i] >= tf[i])
            {
                // regenerate new state

                thrust::generate(&y[i * STATE_DIM], &y[(i + 1) * STATE_DIM] - 1, [&]
                                 { return dist(rng); });
                t[i] = 0.0;
                nstates++;
            }
            // error estimation
            tpe = 0.0;
            for (int j = 0; j < STATE_DIM; j++)
            {
                err_level = ode.getATol() + ode.getRTol() * y[i * STATE_DIM + j];
                err_estimate = abs(y[i * STATE_DIM + j] - y5[i * STATE_DIM + j]);
                tpe = max(tpe, err_estimate / err_level);
            }
            if (tpe >= 1.1)
            {
                // reduce step
                s = max(0.2f, 0.9 * pow(tpe, -0.25f));
            }
            else
            {
                // accept new state

                for (int j = 0; j < STATE_DIM; j++)
                    ys[i * STATE_DIM + j] = y[i * STATE_DIM + j];
                t[i] += step[i];
                // increase step
                s = min(5.0f, 0.9 * pow(tpe, -0.20f));
            }
            s *= step[i];
            step[i] = min(s, tf[i] - step[i]);
        }
    }

    res_file.close();
    return 0;
}